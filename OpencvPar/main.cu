#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <math.h>
#include <IL/il.h>
#include <typeinfo>

using namespace std;

__global__ void copy(unsigned char * mat_in, unsigned char * mat_out, std::size_t cols, std::size_t rows) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;

  //if (i*cols+j < 3*cols*rows) equivalent
  if (j<rows*3 && i<cols)
  {
    mat_out[j * cols*3 + i] = mat_in[j * cols*3 + i];

  }
}

__global__ void blur(unsigned char * mat_in, unsigned char * mat_out, std::size_t cols, std::size_t rows) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
  auto j = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

  //if (j<rows*3 && i<cols && j>3 )
  if (j>2 && j<rows*3 && i<cols)
  {
    //p1 à p9 correspondent aux 9 pixels à récupérer
    unsigned char p1 = mat_in[(j-3) * cols + i - 3];
    unsigned char p2 = mat_in[(j-3) * cols + i];
    unsigned char p3 = mat_in[(j-3) * cols + i + 3];
    unsigned char p4 = mat_in[j * cols + i - 3];
    unsigned char p5 = mat_in[j * cols + i];
    unsigned char p6 = mat_in[j * cols + i + 3];
    unsigned char p7 = mat_in[(j+3) * cols + i - 3];
    unsigned char p8 = mat_in[(j+3) * cols + i];
    unsigned char p9 = mat_in[(j+3) * cols + i + 3];

    mat_out[j * cols + i] = (p1+p2+p3+p4+p5+p6+p7+p8+p9)/9;
  }
  //pour la premiere ligne
  else if (j<=2 && j<rows*3 && i<cols)
  {
    unsigned char p4 = mat_in[j * cols + i - 3];
    unsigned char p5 = mat_in[j * cols + i];
    unsigned char p6 = mat_in[j * cols + i + 3];
    unsigned char p7 = mat_in[(j+3) * cols + i - 3];
    unsigned char p8 = mat_in[(j+3) * cols + i];
    unsigned char p9 = mat_in[(j+3) * cols + i + 3];

    mat_out[j * cols + i] = (p4+p5+p6+p7+p8+p9)/6;
  }
}

__global__ void sharpen(unsigned char * mat_in, unsigned char * mat_out, std::size_t cols, std::size_t rows) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
  auto j = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

  //if (j<rows*3 && i<cols && j>3 )
  if (j>2 && j<rows*3 && i<cols)
  {
    //p1 à p9 correspondent aux 9 pixels à récupérer
    unsigned char p2 = mat_in[(j-3) * cols + i];
    unsigned char p4 = mat_in[j * cols + i - 3];
    unsigned char p5 = mat_in[j * cols + i];
    unsigned char p6 = mat_in[j * cols + i + 3];
    unsigned char p8 = mat_in[(j+3) * cols + i];

    int tmp =  (-3*(p2+p4+p6+p8)+21*p5)/9;
    if (tmp > 255) tmp = 255;
    if (tmp < 0) tmp = 0;
    mat_out[j * cols + i] = tmp;
  }
}

__global__ void edge_detect(unsigned char * mat_in, unsigned char * mat_out, std::size_t cols, std::size_t rows) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x; //pos de la couleur sur x
  auto j = blockIdx.y * blockDim.y + threadIdx.y; //pos de la couleur sur y

  //if (j<rows*3 && i<cols && j>3 )
  if (j>2 && j<rows*3 && i<cols)
  {
    //p1 à p9 correspondent aux 9 pixels à récupérer
    unsigned char p2 = mat_in[(j-3) * cols + i];
    unsigned char p4 = mat_in[j * cols + i - 3];
    unsigned char p5 = mat_in[j * cols + i];
    unsigned char p6 = mat_in[j * cols + i + 3];
    unsigned char p8 = mat_in[(j+3) * cols + i];

    int tmp =  (9*(p2+p4+p6+p8)-36*p5)/9;
    if (tmp > 255) tmp = 255;
    if (tmp < 0) tmp = 0;
    mat_out[j * cols + i] = tmp;
  }
}


int main()
{
  //Declarations
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED);
  unsigned char * rgb = m_in.data;
  int rows = m_in.rows;
  int cols = m_in.cols;

  // cv::Mat planes[3];
  // cv::split(m_in, planes);
  // unsigned char * b = planes[0].data;
  // unsigned char * v = planes[1].data;
  // unsigned char * r = planes[2].data;
  // for (int i=0; i<rows; i++)
  //   for (int j=0; j<cols; j++)
  //     cout << r[j * cols + i];


  vector<unsigned char> g(rows * cols * 3); //Pour recreer l'image
  cv::Mat m_out(rows, cols, CV_8UC3, g.data());
  unsigned char * mat_in;
  unsigned char * mat_out;

  //Init donnes kernel
  hipMalloc( &mat_in, 3 * rows * cols );
  hipMalloc( &mat_out, 3 * rows * cols );
  hipMemcpy( mat_in, rgb, 3 * rows * cols, hipMemcpyHostToDevice );


  //Calcul du nb de blocs et de threads
  // dim3 t( 32, 32 );
  // dim3 b( ( cols - 1) / (t.x-2) + 1 , ( rows - 1 ) / (t.y-2) + 1 );

  dim3 block( 32, 32); //nb de thread, max 1024
  dim3 grid(((cols-1) / block.x + 1), 3*((rows-1) / block.y + 1));

  //Debut de chrono
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  //Appel kernel
  // blur<<< grid, block>>>(mat_in, mat_out, cols, rows);
  // sharpen<<< grid, block>>>(mat_in, mat_out, cols, rows);
  edge_detect<<< grid, block>>>(mat_in, mat_out, cols, rows);

  //Fin de chrono
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cout << hipGetErrorString(hipGetLastError()) << endl;
  float elapsedTime;
  hipEventElapsedTime( &elapsedTime, start, stop);
  cout << elapsedTime << endl;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  //Recup donnees kernel
  hipMemcpy( g.data(), mat_out, 3*rows * cols, hipMemcpyDeviceToHost );

  cv::imwrite( "out.jpg", m_out );

  hipFree(mat_in);
  hipFree(mat_out);


//
//   int cols = 5;
//   int rows = 10;
//   unsigned char * mat;
//   hipMalloc(mat, 3 * rows * cols );
//   for (int i=0; i<cols*rows*3; i++)
//   {
//       mat[i] = i+1;
//   }
//   cv::Mat m = (rows, cols, CV_8UC3, mat);
// cv::imwrite( "test.jpg", m );


  return 0;
}
