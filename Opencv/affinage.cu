#include "hip/hip_runtime.h"
#include ""

#include <opencv2/opencv.hpp>
#include <vector>

using namespace std;

void sharpen(unsigned char* rgb_in, unsigned char* rgb_out, int rows, int cols) {
    for (int row = 1; row < rows - 1; ++row) {
        for (int col = 1; col < cols - 1; ++col) {
            for (int i = 0; i < 3; ++i)
            {
                unsigned char h = rgb_in[3 * (row * cols + col) + i];
                unsigned char g = rgb_in[3 * (row * cols + col - 3) + i];
                unsigned char c = rgb_in[3 * (row * cols + col) + i];
                unsigned char d = rgb_in[3 * (row * cols + col + 3) + i];
                unsigned char b = rgb_in[3 * (row * cols + col) + i];
                // rgb_out[3 * (row * cols + col) + i] = -1 * (h + g + d + b) + 5 * c;
                rgb_out[3 * (row * cols + col) + i] = (-3 * (h + g + d + b) + 21 * c) / 9;
            }
        }
    }
}

int main()
{
    //Declarations
    cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED);
    auto rgb = m_in.data;
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    size_t taille_rgb = 3 * rows * cols;
    std::vector< unsigned char > g(taille_rgb);
    cv::Mat m_out(rows, cols, CV_8UC3, g.data());

    unsigned char* rgb_in;
    unsigned char* rgb_out;

    //Init donnes kernel
    hipHostMalloc(&rgb_in, taille_rgb);
    hipHostMalloc(&rgb_out, taille_rgb);
    hipMemcpy(rgb_in, rgb, taille_rgb, hipMemcpyHostToDevice);

    //Debut de chrono
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    sharpen(rgb_in, rgb_out, rows, cols);

    //Fin de chrono
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    cout << hipGetErrorString(hipGetLastError()) << endl;
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << elapsedTime << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //Recup donnees kernel
    hipMemcpy(g.data(), rgb_out, taille_rgb, hipMemcpyDeviceToHost);
    cv::imwrite("out_sharpen.jpg", m_out);
    hipFree(rgb_in);
    hipFree(rgb_out);
    return 0;
}